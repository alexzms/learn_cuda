#include "hip/hip_runtime.h"
#include "glad/glad.h"
#include "GLFW/glfw3.h"
#include <iostream>
#include "includes/book.h"
#include <chrono>
#include "Shader.h"
#include <cuda_gl_interop.h>

int chapter1();
int chapter2_vector_addition();
int chapter2_julia_set();
float scale = 1.5f; float shift_x = 0.0f; float shift_y = 0.0f; float delta_time = 0.0f;
bool first_mouse = true; float last_x; float last_y; float last_frame = 0.0f; bool need_redraw = true;

const int DIM = 2048;

void process_input(GLFWwindow* window);
void framebuffer_size_callback(GLFWwindow* window, int width, int height);
void scroll_callback(GLFWwindow* window, double unknown_offset, double scroll_offset);
void mouse_callback(GLFWwindow* window, double width_pos, double height_pos);

struct complex_number {
    float r;
    float i;
    // supports both host init and device init
    __host__ __device__ complex_number(float r_, float i_): r(r_), i(i_) {}
    // [[nodiscard]] means that if the return value is not used, there will be a compiler error
    [[nodiscard]]__device__ float magnitude() const { return r * r + i * i; }
    __device__ complex_number operator*(const complex_number &rhs) const {
        return {r * rhs.r - i * rhs.i, r * rhs.i + i * rhs.r};
    }
    __device__ complex_number operator+(const complex_number &rhs) const {
        return {r + rhs.r, i + rhs.i};
    }
};

complex_number C(-0.8f, 0.156f);

__device__ float julia(const int x, const int y, const int DIM, const complex_number C,
                     float scale_dev, float shift_dev_x, float shift_dev_y) {
    // we want it to scale at the center of (shift_dev_x, shift_dev_y)
    float jx = scale_dev * (float)(DIM / 2 - x + shift_dev_x / scale_dev);
    float jy = scale_dev * (float)(DIM / 2 - y + shift_dev_y / scale_dev);

    jx /= DIM / 2;
    jy /= DIM / 2;

    complex_number Z(jx, jy);

    // max iter = 200 (hardcoded)
    for (int i = 0; i != 200; ++i) {
        Z = Z * Z + C;
        if (Z.magnitude() > 4.0f) {
            return (float)i;
        }
    }
    return 0;
}

__global__ void add_integer(int a, int b, int* c) {
    *c = a + b;
}

__global__ void add_vector(const int* a, const int* b, int* c, const int length) {
    int tid = blockIdx.x;
    if (tid < length) {
        c[tid] = a[tid] + b[tid];
    }
}

__global__ void compute_julia(unsigned char* ptr, complex_number C,
                              float scale_dev, float shift_dev_x, float shift_dev_y) {
    int i = blockIdx.x;
    int j = blockIdx.y;

    int offset = i + j * gridDim.x;
    int julia_value = julia(i, j, gridDim.x, C, scale_dev, shift_dev_x, shift_dev_y);
    if (julia_value > 50) {
        julia_value = 50;
    }
    // 3, 4, 5: silver
    ptr[offset * 4 + 0] = 3 * julia_value;
    ptr[offset * 4 + 1] = 4 * julia_value;
    ptr[offset * 4 + 2] = 5 * julia_value;
    ptr[offset * 4 + 3] = 255 * julia_value > 0 ? 255 : 0;
}

__global__ void compute_julia_interop(uchar4* ptr, complex_number C, float scale_dev, float shift_dev_x, float shift_dev_y) {

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    int offset = x + y * gridDim.x * blockDim.x;

    float julia_value = julia(x, y, gridDim.x * blockDim.x, C, scale_dev, shift_dev_x, shift_dev_y);

//    if (julia_value > 50.0f) {
//        julia_value = 50.0f;
//    }
    // 3, 4, 5: silver
    ptr[offset] = make_uchar4(3.01 * julia_value, 3.07 * julia_value, 3.10 * julia_value,
                              255 * (julia_value > 0 ? 1 : 0));
}

int main() {
    chapter2_julia_set();
    return 0;
}

int chapter2_julia_set() {
    const int image_size = DIM * DIM * 4;
    auto* bitmap_sysmem = (unsigned char*)malloc(image_size);
//    unsigned char* dev_map;
    // zero-copy memory
//    HANDLE_ERROR(hipHostAlloc((void**)&zero_copy_map, image_size, hipHostMallocMapped));
//    HANDLE_ERROR(hipHostGetDevicePointer((void**)&dev_map, zero_copy_map, 0));

    // WOW: use a glfw to visualize the result, it's so cool!!!!!!!!!!!!!!!!!!!!!!
    glfwInit();
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 6);
    GLFWmonitor* glfWmonitor = glfwGetPrimaryMonitor();
    if (glfWmonitor == nullptr) {
        std::cout << "Failed to get primary monitor.." << std::endl;
        glfwTerminate();
        return -1;
    }
    const GLFWvidmode *glfWvidmode = glfwGetVideoMode(glfWmonitor);
    unsigned int RENDER_WIDTH = glfWvidmode->width;
    unsigned int RENDER_HEIGHT = glfWvidmode->height;

    GLFWwindow* window = glfwCreateWindow(RENDER_WIDTH, RENDER_HEIGHT,
                                          "Colors", glfWmonitor, nullptr);
    if (window == nullptr) {
        std::cout << "Failed to create glf windows.." << std::endl;
        glfwTerminate();
        return -1;
    }
    glfwMakeContextCurrent(window);
    if (!gladLoadGLLoader((GLADloadproc) glfwGetProcAddress)) {
        std::cout << "Failed to init GLAD loader.." << std::endl;
        glfwTerminate();
        return -1;
    }
    glViewport(0, 0, RENDER_WIDTH, RENDER_HEIGHT);

    glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);
    glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_DISABLED);
    glfwSetScrollCallback(window, scroll_callback);
    glfwSetCursorPosCallback(window, mouse_callback);

    // a rectangle in front of the camera
    float vertices[] = {
            // positions          // texture coords
            1.0f,  1.0f, 0.0f,   1.0f, 1.0f, // top right
            1.0f, -1.0f, 0.0f,   1.0f, 0.0f, // bottom right
            -1.0f, -1.0f, 0.0f,   0.0f, 0.0f, // bottom left
            -1.0f,  1.0f, 0.0f,   0.0f, 1.0f  // top left
    };

    unsigned int indices[] = {
            0, 1, 3, // first triangle
            1, 2, 3  // second triangle
    };

    Shader shader("./shaders/display_texture.vert", "./shaders/display_texture.frag");
    shader.use();
    unsigned int VBO, VAO, EBO;
    // generate vertex array object
    glGenVertexArrays(1, &VAO);
    glBindVertexArray(VAO);
    // generate vertex buffer object
    glGenBuffers(1, &VBO);
    // generate element buffer object
    glGenBuffers(1, &EBO);
    // bind the vertex array object
    glBindVertexArray(VAO);
    // copy vertices array into vertex buffer
    glBindBuffer(GL_ARRAY_BUFFER, VBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);
    // copy indices array into element buffer
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indices), indices, GL_STATIC_DRAW);
    // configure vertex attributes
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 5 * sizeof(float),
                          (void*) nullptr);
    glEnableVertexAttribArray(0); // pos
    glVertexAttribPointer(1, 2, GL_FLOAT, GL_FALSE, 5 * sizeof(float),
                          (void*) (3 * sizeof(float)));
    glEnableVertexAttribArray(1); // texture coords

    // texture
    unsigned int texture;
    glGenTextures(1, &texture);
    glBindTexture(GL_TEXTURE_2D, texture);
    // set the texture wrapping/filtering options (on the currently bound texture object)
    // set texture wrapping to GL_REPEAT (default wrapping method)
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
    // set texture filtering parameters
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    // set texture filtering parameters
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    // use zero-copy memory as the texture
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, DIM, DIM, 0, GL_RGBA, GL_UNSIGNED_BYTE, bitmap_sysmem);

    // 注册 OpenGL 缓冲区到 CUDA 图形资源
    GLuint bufferObj;
    hipGraphicsResource *resource;
    glGenBuffers(1, &bufferObj);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, bufferObj);
    glBufferData(GL_PIXEL_UNPACK_BUFFER, DIM * DIM * sizeof(uchar4), NULL, GL_DYNAMIC_DRAW);
    HANDLE_ERROR(hipGraphicsGLRegisterBuffer(&resource, bufferObj, cudaGraphicsMapFlagsNone));

    uchar4* devPtr;
    size_t  size;
    // 映射资源
    HANDLE_ERROR(hipGraphicsMapResources(1, &resource, NULL));
    HANDLE_ERROR(hipGraphicsResourceGetMappedPointer((void**)&devPtr, &size, resource));

    dim3 blocks(DIM/32, DIM/32);
    dim3 threads(32,32);
    last_frame = (float)glfwGetTime();
    // render loop
    while (!glfwWindowShouldClose(window)) {
        auto current_frame = (float)glfwGetTime();
        delta_time = current_frame - last_frame;
        last_frame = current_frame;
//        std::cout << "FPS: " << 1.0f / delta_time << " Estimated PCIE bandwidth: "
//                  << 3.0f * (float)image_size / delta_time / 1024 / 1024 / 1024 << " GB/s" << std::endl;
        // input
        process_input(window);
        // render
        glClearColor(0.0f, 0.0f, 0.0f, 1.0f); // state-setting function
        glClear(GL_COLOR_BUFFER_BIT); // state-using function

        // memset the zero-copy memory
        compute_julia_interop<<<blocks, threads>>>(devPtr, C, scale, shift_x, shift_y);
        HANDLE_ERROR(hipDeviceSynchronize());

        // draw the rectangle
        glBindBuffer(GL_PIXEL_UNPACK_BUFFER, bufferObj);
        glBindTexture(GL_TEXTURE_2D, texture);
        glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, DIM, DIM, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
        shader.use();
        glBindVertexArray(VAO);
        glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, nullptr);

        // glfw: swap buffers and poll IO events (keys pressed/released, mouse moved etc.)
        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    HANDLE_ERROR(hipGraphicsUnmapResources(1, &resource, NULL));
    HANDLE_ERROR(hipGraphicsUnregisterResource(resource));
    // glfw: terminate, clearing all previously allocated GLFW resources.
    free(bitmap_sysmem);
    glfwTerminate();
    return 0;
}

/*
 * This chapter is also about the basic usage of cuda, we parallelized the vector addition.
 * And observed the performance.
 */
int chapter2_vector_addition() {
    const unsigned int ARRAY_LENGTH = 65535;
    bool identical = true;
    int *a = (int*)malloc(ARRAY_LENGTH * sizeof(int));
    int *b = (int*)malloc(ARRAY_LENGTH * sizeof(int));
    int *c = (int*)malloc(ARRAY_LENGTH * sizeof(int));
    int *cpu_temp_c = (int*)malloc(ARRAY_LENGTH * sizeof(int));
    int *dev_a, *dev_b, *dev_c;
    // first we assign values to a[] and b[]
    for (int i = 0; i != ARRAY_LENGTH; ++i) {
        a[i] = i;
        b[i] = -2 * i + i % 5;
    }

    auto start_cpu = std::chrono::high_resolution_clock::now();
    for (int i = 0; i != ARRAY_LENGTH; ++i) {
        cpu_temp_c[i] = a[i] + b[i];
    }
    auto end_cpu = std::chrono::high_resolution_clock::now();


    auto start_host_to_device = std::chrono::high_resolution_clock::now();
    // allocate on gpu
    HANDLE_ERROR(hipMalloc((void**)&dev_a, ARRAY_LENGTH * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, ARRAY_LENGTH * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, ARRAY_LENGTH * sizeof(int)));
    // copy to gpu
    HANDLE_ERROR(hipMemcpy(dev_a, a, ARRAY_LENGTH * sizeof(int),
                            hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, ARRAY_LENGTH * sizeof(int),
                            hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_c, c, ARRAY_LENGTH * sizeof(int),
                            hipMemcpyHostToDevice));
    auto end_host_to_device = std::chrono::high_resolution_clock::now();

    auto start_gpu = std::chrono::high_resolution_clock::now();
    add_vector<<<ARRAY_LENGTH, 1>>>
            (dev_a, dev_b, dev_c, ARRAY_LENGTH);
    HANDLE_ERROR(hipDeviceSynchronize());
    auto end_gpu = std::chrono::high_resolution_clock::now();

    auto start_device_to_host = std::chrono::high_resolution_clock::now();
    HANDLE_ERROR(hipMemcpy(c, dev_c, ARRAY_LENGTH * sizeof(int),
                            hipMemcpyDeviceToHost));
    auto end_device_to_host = std::chrono::high_resolution_clock::now();



    // check if the result is correct
    for (int i = 0; i != ARRAY_LENGTH; ++i) {
        if (c[i] != cpu_temp_c[i]) {
            identical = false;
            break;
        }
    }
    if (identical) {
        std::cout << "The result is correct" << std::endl;
    } else {
        std::cout << "The result is incorrect" << std::endl;
    }

    std::cout << "CPU time: " << std::chrono::duration_cast<std::chrono::microseconds>
            (end_cpu - start_cpu).count() << " us" << std::endl;
    std::cout << "GPU time: " << std::chrono::duration_cast<std::chrono::microseconds>
            (end_gpu - start_gpu).count() << " us" << "+"
              << std::chrono::duration_cast<std::chrono::microseconds>
                      (end_host_to_device - start_host_to_device).count() << " us(host->device memcpy)"
              << "+" << std::chrono::duration_cast<std::chrono::microseconds>
                      (end_device_to_host - start_device_to_host).count() << " us(device->host memcpy)"
              << std::endl;
    std::cout << "Hint: the copy time is really large, and gpu time is also not fascinating."
                 "We need to optimize this deeper." << std::endl;

    // free memory
    free(a);
    free(b);
    free(c);
    free(cpu_temp_c);
    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipFree(dev_b));
    HANDLE_ERROR(hipFree(dev_c));
    return 0;
}

/*
 * This chapter is about the basic usage of cuda, including:
 * 1. hipGetDeviceCount
 * 2. hipGetDeviceProperties
 * 3. hipMalloc
 * 4. hipFree
 * 5. hipMemcpy
 * 6. kernel function
 */
int chapter1() {
    int device_count;
    HANDLE_ERROR(hipGetDeviceCount(&device_count));
    std::cout << "device count: " << device_count << std::endl;
    for (int i = 0; i != device_count; ++i) {
        std::cout << "Fetching device properties for device " << i << std::endl;
        hipDeviceProp_t device_prop{};
        HANDLE_ERROR(hipGetDeviceProperties(&device_prop, i));
        std::cout << "Device name: " << device_prop.name << std::endl;
        std::cout << "Compute capability: " << device_prop.major << "." << device_prop.minor << std::endl;
        std::cout << "Total global memory: " << device_prop.totalGlobalMem << std::endl;
        std::cout << "Shared memory per block: " << device_prop.sharedMemPerBlock << std::endl;
        std::cout << "Registers per block: " << device_prop.regsPerBlock << std::endl;
        std::cout << "Warp size: " << device_prop.warpSize << std::endl;
        std::cout << "Memory pitch: " << device_prop.memPitch << std::endl;
        std::cout << "Max threads per block: " << device_prop.maxThreadsPerBlock << std::endl;
        std::cout << "Max threads dimensions: " << device_prop.maxThreadsDim[0] << " " << device_prop.maxThreadsDim[1] << " " << device_prop.maxThreadsDim[2] << std::endl;
        std::cout << "Max grid size: " << device_prop.maxGridSize[0] << " " << device_prop.maxGridSize[1] << " " << device_prop.maxGridSize[2] << std::endl;
        std::cout << "Clock rate: " << device_prop.clockRate << std::endl;
        std::cout << "Total constant memory: " << device_prop.totalConstMem << std::endl;
        std::cout << "Texture alignment: " << device_prop.textureAlignment << std::endl;
        std::cout << "Concurrent copy and execution: " << device_prop.deviceOverlap << std::endl;
        std::cout << "Number of multiprocessors: " << device_prop.multiProcessorCount << std::endl;
        std::cout << "Kernel execution timeout enabled: " << device_prop.kernelExecTimeoutEnabled << std::endl;
        std::cout << "Integrated: " << device_prop.integrated << std::endl;
        std::cout << "Can map host memory: " << device_prop.canMapHostMemory << std::endl;
        std::cout << "Supports page-locked memory: " << device_prop.canMapHostMemory << std::endl;
        std::cout << "Compute mode: " << device_prop.computeMode << std::endl;
        std::cout << "Maximum 1D texture size: " << device_prop.maxTexture1D << std::endl;
        std::cout << "Maximum 2D texture size: " << device_prop.maxTexture2D[0] << " " << device_prop.maxTexture2D[1] << std::endl;
        std::cout << "Maximum 3D texture size: " << device_prop.maxTexture3D[0] << " " << device_prop.maxTexture3D[1] << " " << device_prop.maxTexture3D[2] << std::endl;
        std::cout << "Maximum 1D layered texture dimensions: " << device_prop.maxTexture1DLayered[0] << " " << device_prop.maxTexture1DLayered[1] << std::endl;
        std::cout << "Maximum 2D layered texture dimensions: " << device_prop.maxTexture2DLayered[0] << " " << device_prop.maxTexture2DLayered[1] << " " << device_prop.maxTexture2DLayered[2] << std::endl;
        std::cout << "Surface alignment: " << device_prop.surfaceAlignment << std::endl;
        std::cout << "Concurrent kernels: " << device_prop.concurrentKernels << std::endl;
        std::cout << "ECC enabled: " << device_prop.ECCEnabled << std::endl;
        std::cout << "PCI bus ID: " << device_prop.pciBusID << std::endl;
        std::cout << "PCI device ID: " << device_prop.pciDeviceID << std::endl;
        std::cout << "PCI domain ID: " << device_prop.pciDomainID << std::endl;
        std::cout << "TCC driver: " << device_prop.tccDriver << std::endl;
        std::cout << "Async engine count: " << device_prop.asyncEngineCount << std::endl;
        std::cout << "Unified addressing: " << device_prop.unifiedAddressing << std::endl;
        std::cout << "Memory clock rate: " << device_prop.memoryClockRate << std::endl;
        std::cout << "Memory bus width: " << device_prop.memoryBusWidth << std::endl;
        std::cout << "L2 cache size: " << device_prop.l2CacheSize << std::endl;
        std::cout << "Max threads per multiprocessor: " << device_prop.maxThreadsPerMultiProcessor << std::endl;
        std::cout << "Stream priorities: " << device_prop.streamPrioritiesSupported << std::endl;
        std::cout << "Global L1 cache supported: " << device_prop.globalL1CacheSupported << std::endl;
        std::cout << "Local L1 cache supported: " << device_prop.localL1CacheSupported << std::endl;
        std::cout << "Shared memory per multiprocessor: " << device_prop.sharedMemPerMultiprocessor << std::endl;
        std::cout << "Registers per multiprocessor: " << device_prop.regsPerMultiprocessor << std::endl;
        std::cout << "Managed memory: " << device_prop.managedMemory << std::endl;
        std::cout << "Is multi-GPU board: " << device_prop.isMultiGpuBoard << std::endl;
        std::cout << "Multi-GPU board group ID: " << device_prop.multiGpuBoardGroupID << std::endl;
        std::cout << "Host native atomic supported: " << device_prop.hostNativeAtomicSupported << std::endl;
        std::cout << "Single to double precision perf ratio: " << device_prop.singleToDoublePrecisionPerfRatio << std::endl;
        std::cout << "Pageable memory access: " << device_prop.pageableMemoryAccess << std::endl;
        std::cout << "Concurrent managed access: " << device_prop.concurrentManagedAccess << std::endl;
        std::cout << "Compute preemption supported: " << device_prop.computePreemptionSupported << std::endl;
        std::cout << "Can use host pointer for registered memory: " << device_prop.canUseHostPointerForRegisteredMem << std::endl;
        std::cout << "Cooperative launch: " << device_prop.cooperativeLaunch << std::endl;
        std::cout << "Cooperative multi-device launch: " << device_prop.cooperativeMultiDeviceLaunch << std::endl;
        std::cout << "Pageable memory access uses host page tables: " << device_prop.pageableMemoryAccessUsesHostPageTables << std::endl;
        std::cout << "Direct managed memory access from host: " << device_prop.directManagedMemAccessFromHost << std::endl;
        std::cout << "Max blocks per multiprocessor: " << device_prop.maxBlocksPerMultiProcessor << std::endl;
    }
    int device_id;
    HANDLE_ERROR(hipGetDevice(&device_id));
    std::cout << "Current device id: " << device_id << std::endl;
    hipDeviceProp_t ideal_device_prop{};
    memset(&ideal_device_prop, 0, sizeof(hipDeviceProp_t));
    ideal_device_prop.major = 1;
    ideal_device_prop.minor = 3;
    HANDLE_ERROR(hipChooseDevice(&device_id, &ideal_device_prop));
    std::cout << "Ideal device id: " << device_id << std::endl;
    HANDLE_ERROR(hipSetDevice(device_id));
    std::cout << "Setting current device id to " << device_id << std::endl;

    int* result = (int*)malloc(sizeof(int));
    int* dev_result;
    // HANDLE_ERROR: simple implementation of error catch
    // hipMalloc: Just like malloc, but the parameter is pointer of pointer
    HANDLE_ERROR(hipMalloc((void**)&dev_result, sizeof(int)));

    add_integer<<<1, 1>>>(3, 4, dev_result);
    // the following code is not allowed, because modifying cuda memory out of add_integer function
    // will result in memory error
    // *dev_result = 5;

    // copy the cuda memory from device gpu to host system dram
    HANDLE_ERROR(
            hipMemcpy(result, dev_result, sizeof(int), hipMemcpyDeviceToHost)
    );

    std::cout << "Hello, World! result=" << *result << std::endl;
    hipFree(dev_result);
    free(result);
    return 0;
}

void framebuffer_size_callback(GLFWwindow* window, int width, int height) {
    need_redraw = true;
    glViewport(0, 0, width, height);
}

void process_input(GLFWwindow* window) {
    // glfwGetKey: get the state of the key
    if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS) {
        glfwSetWindowShouldClose(window, true);
    }
    // wasd
    float pixel_per_input = 30 * scale;
    if (glfwGetKey(window, GLFW_KEY_W) == GLFW_PRESS) {
        shift_y -= pixel_per_input;
    }
    if (glfwGetKey(window, GLFW_KEY_S) == GLFW_PRESS) {
        shift_y += pixel_per_input;
    }
    if (glfwGetKey(window, GLFW_KEY_A) == GLFW_PRESS) {
        shift_x += pixel_per_input;
    }
    if (glfwGetKey(window, GLFW_KEY_D) == GLFW_PRESS) {
        shift_x -= pixel_per_input;
    }
    // q, e
    if (glfwGetKey(window, GLFW_KEY_Q) == GLFW_PRESS) {
        scale *= 1.0f + 0.7f * delta_time;
    }
    if (glfwGetKey(window, GLFW_KEY_E) == GLFW_PRESS) {
        scale /= 1.0f + 0.7f * delta_time;
    }
    // z to increase real part of C, x to decrease real part of C
    if (glfwGetKey(window, GLFW_KEY_Z) == GLFW_PRESS) {
        C.r += 0.0001f;
    }
    if (glfwGetKey(window, GLFW_KEY_X) == GLFW_PRESS) {
        C.r -= 0.0001f;
    }
    // c to increase imaginary part of C, v to decrease imaginary part of C
    if (glfwGetKey(window, GLFW_KEY_C) == GLFW_PRESS) {
        C.i += 0.0001f;
    }
    if (glfwGetKey(window, GLFW_KEY_V) == GLFW_PRESS) {
        C.i -= 0.0001f;
    }
}

void scroll_callback(GLFWwindow* window, double unknown_offset, double scroll_offset) {
    need_redraw = true;
    const float coefficient = 0.1f * scale;
    scale -= coefficient * (float)scroll_offset;
    if (scale <= 0.0f) {
        scale = 0.00001f;
    }
//    std::cout << "scale: " << scale << std::endl;
}

void mouse_callback(GLFWwindow* window, double width_pos, double height_pos) {
    if (first_mouse) {
        last_x = width_pos;
        last_y = height_pos;
        first_mouse = false;
    }
    float x_offset = width_pos - last_x;
    float y_offset = last_y - height_pos;
    last_x = width_pos;
    last_y = height_pos;
    const float sensitivity = delta_time * 100.0f * scale;
    x_offset *= sensitivity;
    y_offset *= sensitivity;
    shift_x -= x_offset;
    shift_y -= y_offset;
//    std::cout << "shift_x: " << shift_x << " shift_y: " << shift_y << std::endl;
}