#include "hip/hip_runtime.h"
/*
 * This program should've use texture memory to speed up the data fetching, but texture reference is deprecated!!!
 * No more work on this... ABANDONED
 */

#include <hip/hip_runtime.h>
#include "hip/hip_texture_types.h"
#include "iostream"
#include "random"
#include <cstdio>
#include "./includes/book.h"
#include "./includes/cpu_anim.h"
#include "check_error.cuh"
#include <cstdio>

#define DIM 1024
#define FLOW_SPEED 0.25f
#define ANIMATION_SPEED 90
#define MAX_TEMP 1.0f
#define MIN_TEMP 0.0001f


struct DataBlock {
    unsigned char *d_output_bitmap;
    float *d_iptr;
    float *d_optr;
    float *d_sptr;
    CPUAnimBitmap *h_bitmap;

    hipEvent_t start, stop;
    float total_time;
    unsigned int frames_count;
};

__global__ void maintain_source_constant_temperature_kernel(float *iptr, const float *sptr) {
    unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;

    unsigned int offset = x + y * gridDim.x * blockDim.x;

    // if sptr[i] is not zero, set iptr[i] to sptr[i] to maintain the source temperature
    // we can compare to 0 safely because value of sptr will be written as 0, this won't cause problem
    if (sptr[offset] != 0) {
        iptr[offset] = sptr[offset];
    }
}

// instead of directly update the iptr, we will output result to optr
// because if we do it on iptr, the result value will depend on the order of execution of code, which is not ideal
__global__ void heat_flow_kernel(float *optr, const float *iptr) {
    unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;

    unsigned int offset = x + y * gridDim.x * blockDim.x;

    unsigned int left_offset = offset - 1;
    unsigned int right_offset = offset + 1;
    // prevent overflow, the reason we still keep this is because later, that term will cancel out
    // like when x == 0, left == 0, ptr[left] = ptr[x], so T_new = ... T_left + ... - 4T_old, they cancel out
    if (x == 0) {
        left_offset += 1;
    } else if (x == DIM - 1) {
        right_offset -= 1;
    }
    unsigned int up_offset = offset - DIM;
    unsigned int down_offset = offset + DIM;
    if (y == 0) {
        up_offset += DIM;
    } else if (y == DIM - 1) {
        down_offset -= DIM;
    }
    optr[offset] = iptr[offset]
         + FLOW_SPEED * (iptr[left_offset]+iptr[right_offset]+iptr[up_offset]+iptr[down_offset] - 4 * iptr[offset]);
}

__global__ void float_to_color_kernel(unsigned char *obitmap, const float *iptr) {
    unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;

    unsigned int offset = x + y * gridDim.x * blockDim.x;

    float value = iptr[offset];
    if (value > MAX_TEMP) {
        value = MAX_TEMP;
    }
    if (value < MIN_TEMP) {
        value = MIN_TEMP;
    }
    // when it's MAX_TEMP, it's red, when it's MIN_TEMP, it's blue, we will do a linear interpolation
    // to get the color value
    float color_value = (value - MIN_TEMP) / (MAX_TEMP - MIN_TEMP);
    obitmap[offset * 4 + 0] = (unsigned char)(255 * color_value);
    obitmap[offset * 4 + 1] = 0;
    obitmap[offset * 4 + 2] = (unsigned char)(255 * (1 - color_value));
    obitmap[offset * 4 + 3] = 255;
}

void anim_gpu(DataBlock *data_block, int tick) {
    CHECK_ERROR(hipEventRecord(data_block->start, nullptr));
    dim3 grid(DIM/16, DIM/16);
    dim3 block(16, 16);
    CPUAnimBitmap *h_bitmap = data_block->h_bitmap;
    for (int i = 0; i != ANIMATION_SPEED; ++i) {
        maintain_source_constant_temperature_kernel<<<grid, block>>>
                                                        (data_block->d_iptr, data_block->d_sptr);
        heat_flow_kernel<<<grid, block>>>(data_block->d_optr, data_block->d_iptr);
        // swap
//        float *d_temp = data_block->d_iptr;
//        data_block->d_iptr = data_block->d_optr;
//        data_block->d_optr = d_temp;
        std::swap(data_block->d_iptr, data_block->d_optr);
    }
    float_to_color<<<grid, block>>>
                                                (data_block->d_output_bitmap, data_block->d_iptr);
    CHECK_ERROR(hipMemcpy(h_bitmap->get_ptr(), data_block->d_output_bitmap,
                           h_bitmap->image_size(), hipMemcpyDeviceToHost));
    float elapsed_time = 0.0f;
    CHECK_ERROR(hipEventRecord(data_block->stop, nullptr));
    CHECK_ERROR(hipEventSynchronize(data_block->stop));
    CHECK_ERROR(hipEventElapsedTime(&elapsed_time, data_block->start, data_block->stop));
    data_block->total_time += elapsed_time;
    data_block->frames_count += 1;
    printf("[INFO] Tick: %u, FPS(current): %3.1f, FPS(average): %3.1f\n",
           tick, 1000.0f / elapsed_time, 1000.0f / (data_block->total_time / data_block->frames_count));
}

void anim_exit_gpu(DataBlock *data_block) {
    CHECK_ERROR(hipFree(data_block->d_iptr));
    CHECK_ERROR(hipFree(data_block->d_optr));
    CHECK_ERROR(hipFree(data_block->d_sptr));
    CHECK_ERROR(hipEventDestroy(data_block->start));
    CHECK_ERROR(hipEventDestroy(data_block->stop));
}

int main() {
    DataBlock data_block{};
    CPUAnimBitmap bitmap(DIM, DIM, &data_block);
    data_block.h_bitmap = &bitmap;
    data_block.total_time = 0.0f;
    data_block.frames_count = 0;

    CHECK_ERROR(hipEventCreate(&data_block.start));
    CHECK_ERROR(hipEventCreate(&data_block.stop));
    CHECK_ERROR(hipMalloc((void**)&data_block.d_output_bitmap, bitmap.image_size()));
    unsigned int float_bitmap_size = DIM * DIM * sizeof(float);
    CHECK_ERROR(hipMalloc((void**)&data_block.d_iptr, float_bitmap_size));
    CHECK_ERROR(hipMalloc((void**)&data_block.d_optr, float_bitmap_size));
    CHECK_ERROR(hipMalloc((void**)&data_block.d_sptr, float_bitmap_size));

    auto *h_sptr = new float[DIM * DIM];
    for (int i = 0; i != DIM * DIM; ++i) {
        h_sptr[i] = 0.0f;
        unsigned int x = i % DIM;
        unsigned int y = i / DIM;
        if ((x > 300) && (x < DIM-300) && (y > 310) && (y < 601)) {
            h_sptr[i] = MAX_TEMP;
        }
    }
    h_sptr[DIM * 100 + 100] = (MAX_TEMP + MIN_TEMP) / 2;
    h_sptr[DIM * 700 + 100] = MIN_TEMP;
    h_sptr[DIM * 300 + 300] = MIN_TEMP;
    h_sptr[DIM * 200 + 700] = MIN_TEMP;
    for (int y = 800; y != 900; ++y) {
        for (int x = 400; x != DIM-400; ++x) {
            h_sptr[x + y * DIM] = MIN_TEMP;
        }
    }

    for (int y = 100; y != 200; ++y) {
        for (int x = 400; x != DIM-400; ++x) {
            h_sptr[x + y * DIM] = MIN_TEMP;
        }
    }
    CHECK_ERROR(hipMemcpy(data_block.d_sptr, h_sptr, float_bitmap_size, hipMemcpyHostToDevice));
    for (int y = 800; y != DIM - 100; ++y) {
        for (int x = 100; x != 200; ++x) {
            h_sptr[x + y * DIM] = MAX_TEMP;
        }
    }
    CHECK_ERROR(hipMemcpy(data_block.d_iptr, h_sptr, float_bitmap_size, hipMemcpyHostToDevice));
    delete[] h_sptr;

    bitmap.anim_and_exit((void (*)(void *, int))anim_gpu, (void (*)(void *))anim_exit_gpu);
    CHECK_ERROR(hipFree(data_block.d_output_bitmap));
    CHECK_ERROR(hipFree(data_block.d_iptr));
    CHECK_ERROR(hipFree(data_block.d_optr));
    CHECK_ERROR(hipFree(data_block.d_sptr));
    return 0;
}